#include "hip/hip_runtime.h"
#include "Simulator.h"

__device__ float calcGainTransmittance(float distance)
{
    return expf(-0.26 * distance / 1000);
}

__device__ float norm(float2 input)
{
    return sqrtf(input.x * input.x + input.y * input.y);
}

__device__ float inner_product(float2 a, float2 b)
{
    return (a.x * b.x + a.y * b.y);
}


__device__ bool isInsideObject(float2* data, int length_data, float2 imgPos)
{
    float sum = 0;
    for(int i = 0; i < length_data; i++)
    {
        float2 vector1;
        float2 vector2;

        vector1.x = data[i].x - imgPos.x;
        vector1.y = data[i].y - imgPos.y;
        if(i == length_data - 1)
        {
            vector2.x = data[0].x - imgPos.x;
            vector2.y = data[0].y - imgPos.y;
        }
        else
        {
            vector2.x = data[i + 1].x - imgPos.x;
            vector2.y = data[i + 1].y - imgPos.y;
        }

        float cos_phi = inner_product(vector1, vector2) / norm(vector1) / norm(vector2);
        sum = sum + acosf(cos_phi);
    }

    if(sum > 2 * M_PI - 0.001)
    {
        return true;
    }
    else
    {
        return false;
    }
}

__device__ float calcDistanceToFaceHelper()
{

}

__device__ float2 cudaImageModel(ObjStatus missile, GPS target_gps,
                                 float * Rb2c_cur, float *Ri2b_missile_cur,
                                 float * Re2i_missile, float fov_pixel)
{
    float2 result;
    Coordinate target_pos = Geoditic2ECEF(target_gps);
    Coordinate missile_pos = Geoditic2ECEF(missile.gps);
    float distance = sqrtf((missile_pos.x - target_pos.x) * (missile_pos.x - target_pos.x) +
                           (missile_pos.y - target_pos.y) * (missile_pos.y - target_pos.y) +
                           (missile_pos.z - target_pos.z) * (missile_pos.z - target_pos.z));

    float NED[3];
    NED[0] = (missile_pos.x - target_pos.x) / distance;
    NED[1] = (missile_pos.y - target_pos.y) / distance;
    NED[2] = (missile_pos.z - target_pos.z) / distance;

    float Ldonic[3];
    float temp1[9];
    float temp2[9];
    mul3x3TransposeBoth(&temp1[0], Rb2c_cur, Ri2b_missile_cur);
    mul3x3(&temp2[0], &temp1[0], Re2i_missile);
    mul3x3ToVec3x1(&Ldonic[0], &temp2[0], &NED[0]);
    if(Ldonic[2] < 0)
    {
        Ldonic[2] = - Ldonic[2];
    }
    result.x = Ldonic[0] / Ldonic[2] * fov_pixel;
    result.y = Ldonic[1] / Ldonic[2] * fov_pixel;
    return  result;
}

__global__ void cudaConvertToImage(GPS * ship_gps, float2 * shipImgPos,
                                   float3 * ship_vertices, ObjStatus missile, ObjStatus target,
                                   float * Ri2b_target, float * Re2i_target,
                                   int num_vertices)
{
    int idx = threadIdx.x + IMUL(blockDim.x, blockIdx.x);
    if(idx < num_vertices)
    {
        float vertex[3];
        vertex[0] = ship_vertices[idx].x;
        vertex[1] = ship_vertices[idx].y;
        vertex[2] = ship_vertices[idx].z;

        float temp[9];
        float NED[3];
        mul3x3TransposeFirst(&temp[0], Re2i_target, Ri2b_target);
//        mul3x3ToVec3x1(&NED[0], temp, vertex);
//        Coordinate tmp = Geoditic2ECEF(target.gps) + Coordinate(NED[0], NED[1], NED[2]);
    }
}

void Simulator::convertToImage(ShipInfo &ship, ObjStatus &missile, ObjStatus &target)
{
    Coordinate target_pos = Geoditic2ECEF(target.gps);
    cv::Mat Ri2b_target(3, 3, CV_32FC1, m_Ri2b_target);
    cv::Mat Re2i_target(3, 3, CV_32FC1, m_Re2i_target);

    for(int i = 0; i < ship.num_vertices; i++)
    {
        cv::Mat vertex(3, 1, CV_32FC1, (float*)(ship.vertices + i));
        cv::Mat NED(3, 1, CV_32FC1);
        NED = Re2i_target.t() * Ri2b_target * vertex;
        Coordinate temp;
        temp.x = target_pos.x + NED.at<float>(0,0);
        temp.y = target_pos.y + NED.at<float>(1,0);
        temp.z = target_pos.z + NED.at<float>(2,0);
        ship.gps[i] = ECEF2Geoditic(temp);
        ship.imgPos[i] = imageModel(missile, ship.gps[i]);
    }
}

__global__ void cudaCalcDistanceToFace(float *distance,
                                       int num_faces, int num_partialPix, int offset,
                                       int batch_size, int width, int height)
{
    int partialPixIdx = threadIdx.x + IMUL(blockIdx.x, blockDim.x);
    int faceIdx = threadIdx.y + IMUL(blockIdx.y, blockDim.y);
    int resultIdx = faceIdx * batch_size * PIXEL_GRID_SIZE * PIXEL_GRID_SIZE + partialPixIdx;

    int pixIdx = offset * batch_size + partialPixIdx / (PIXEL_GRID_SIZE * PIXEL_GRID_SIZE);
    int row = pixIdx / width;
    int col = pixIdx % width;
    int partialPixIdx_X = partialPixIdx / (batch_size * PIXEL_GRID_SIZE);
    int partialPixIdx_Y = partialPixIdx % (batch_size * PIXEL_GRID_SIZE);

    if(faceIdx < num_faces && pixIdx < num_partialPix)
    {
        float u = col + (float)(partialPixIdx_X - 0.5)/ PIXEL_GRID_SIZE - width / 2 - 0.5;
        float w = row + (float)(partialPixIdx_Y - 0.5)/ PIXEL_GRID_SIZE - height / 2 - 0.5;
//        if(isInsideObject())
//        {

//        }
//        else
//        {
//            distance[resultIdx] = 0;
//        }
    }
}


void Simulator::gpuCalcDistanceToFace()
{
    dim3 blockDim(threadsPerBlock, threadsPerBlock);
    dim3 gridDim(ceil((float)m_batch_size * PIXEL_GRID_SIZE * PIXEL_GRID_SIZE / threadsPerBlock),
                 ceil((float)m_ship.num_surfaces / threadsPerBlock));


    gpuErrChk(hipDeviceSynchronize());
}


void Simulator::calcTranformationMatrices()
{
    ObjStatus missile_cur = m_missile[m_current_img_id];
    ObjStatus target_cur = m_target[m_current_img_id];
    SeekerInfo seeker_cur = m_seeker[m_current_img_id];
    ObjStatus missile_prev;
    ObjStatus target_prev;
    SeekerInfo seeker_prev;

    if(m_current_img_id > 0)
    {
        missile_prev = m_missile[m_current_img_id - 1];
        target_prev = m_target[m_current_img_id - 1];
        seeker_prev = m_seeker[m_current_img_id - 1];
    }
    else {
        missile_prev = m_missile[m_current_img_id];
        target_prev = m_target[m_current_img_id];
        seeker_prev = m_seeker[m_current_img_id];
    }

    getRb2cMatrix(m_Rb2c_cur, RotationAngle(0, seeker_cur.elevation, seeker_cur.azimuth));
    getRb2cMatrix(m_Rb2c_prev, RotationAngle(0, seeker_prev.elevation, seeker_prev.azimuth));
    getRi2bMatrix(m_Ri2b_missile_cur, missile_cur.angle);
    getRi2bMatrix(m_Ri2b_missile_prev, missile_prev.angle);
    getRi2bMatrix(m_Ri2b_target, target_cur.angle);
    getRe2iMatrix(m_Re2i_missile, missile_cur.gps);
    getRe2iMatrix(m_Re2i_target, target_cur.gps);
}
